#include "hip/hip_runtime.h"
#include "gpu/gpu-diag-gmm.h"
#include <algorithm>

#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>

#include <math_functions.h>
#include <hip/hip_math_constants.h>

namespace kaldi{

__device__
int32 GPUDiagGmm::Dim() const { return means_invvars_.NumCols(); }

GPUDiagGmm::GPUDiagGmm(DiagGmm &d):
  valid_gconsts_(d.valid_gconsts()),
  gconsts_(d.gconsts()),
  weights_(d.weights()),
  inv_vars_(d.inv_vars()),
  means_invvars_(d.means_invvars()) {}

// TODO : Implement this!
__device__ BaseFloat GPUDiagGmm::LogLikelihood(BaseFloat *data, int32 num_data){
  const double kGPUMinLogDiffDouble = log(DBL_EPSILON);
  const float kGPUMinLogDiffFloat = log(FLT_EPSILON);

  // if (!valid_gconsts_)
  //   KALDI_ERR << "Must call ComputeGconsts() before computing likelihood";

  /* BEGIN LogLikelihoods */
  int32 num_loglikes = gconsts_.Dim();
  BaseFloat* loglikes = new BaseFloat[num_loglikes];
  for(int32 i = 0;i < num_loglikes; ++i) loglikes[i] = gconsts_.data[i];

  // if (num_data != Dim()) {
  //   KALDI_ERR << "DiagGmm::ComponentLogLikelihood, dimension "
  //             << "mismatch " << num_data << " vs. "<< Dim();
  // }

  BaseFloat* data_sq = new BaseFloat[num_data];
  for(int32 i = 0;i < num_data; ++i) data_sq[i] = data[i] * data[i];

  for(int i = 0;i < gconsts_.Dim(); ++i){
    for(int j = 0;j < num_data; ++j){
      loglikes[i] += means_invvars_.data[means_invvars_.Index(i, j)] * data[j];
      loglikes[i] -= 0.5 * inv_vars_.data[inv_vars_.Index(i, j)] * data_sq[j];
    }
  }

  /* END LogLikelihoods */

  /* Begin Log Sum Exp */
  BaseFloat max_elem = (sizeof(BaseFloat) == 4) ? HIP_MIN_DENORM_F : HIP_MIN_DENORM;
  for(int32 i = 0;i < num_loglikes; ++i) {
    if(max_elem < loglikes[i]) max_elem = loglikes[i];
  }

  BaseFloat cutoff;
  if (sizeof(BaseFloat) == 4) cutoff = max_elem + kGPUMinLogDiffFloat;
  else cutoff = max_elem + kGPUMinLogDiffDouble;
  double sum_relto_max_elem = 0.0;

  for (int32 i = 0; i < num_loglikes; i++) {
    BaseFloat f = loglikes[i];
    if (f >= cutoff)
      sum_relto_max_elem += exp(f - max_elem);
  }
  BaseFloat log_sum = max_elem + log(sum_relto_max_elem);
  /* End Log Sum Exp */

  // if (isnan(log_sum) || isinf(log_sum))
  //   KALDI_ERR << "Invalid answer (overflow or invalid variances/features?)";

  delete [] loglikes;
  delete [] data_sq;
  return log_sum;
}

}
